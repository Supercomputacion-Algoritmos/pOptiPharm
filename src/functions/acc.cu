#include "hip/hip_runtime.h"
#include "functions/acc.cuh"

#include "functions/Tanimoto.h"
#include "uego/uego.h"

Acc::Acc() {
  this->use_cuda = false;

  this->query_atoms = nullptr;
  this->query_weight = nullptr;
  this->query_radius = nullptr;

  this->target_atoms = nullptr;
  this->target_weight = nullptr;
  this->target_radius = nullptr;
}

void Acc::initialize(Ini *ini) {
  // Query CUDA props.
  int max_devs;
  CUDA_CHECK(hipGetDeviceCount(&max_devs));

  this->max_CUDA_devices = (uint32_t)max_devs;
  this->max_threads = ini->getThreadNumber();

  uint32_t query_size = ini->getMolQuery()->getNumAtoms();
  uint32_t target_size = ini->getMolVariable()->getNumAtoms();
  uint32_t combined_size = query_size + target_size;

  if (ini->isForcingCUDA() || combined_size >= ini->getCUDAMinCombinedSize()) {
    this->use_cuda = true;

    printf("cuda     : -- GPU support enabled. Molecules combined size (%d, "
           "Query: %d + Target: %d) greater or equal to limit (%d).CPU\n",
           combined_size, query_size, target_size,
           ini->getCUDAMinCombinedSize());

    if (this->max_threads > this->max_CUDA_devices) {
      printf("cuda     : -- There are more CPU threads (%d) than CUDA devices "
             "(%d). "
             "Some CPUthreads will use the same CUDA device. \n"
             "              It MAY incur in a performance penalty. Keep "
             "watching!\n",
             this->max_threads, this->max_CUDA_devices);
    }

    for (uint32_t i = 0; i < this->max_CUDA_devices; i++) {
      hipDeviceProp_t props;
      CUDA_CHECK(hipGetDeviceProperties(&props, i));

      printf("cuda     : -- Using CUDA device %d: %s - %.2lf GiB\n", i,
             props.name,
             ceil(props.totalGlobalMem / 1024.0f / 1024.0f / 1024.0f));
    }
  } else {
    printf("cuda     : -- GPU support disabled. Molecules combined size (%d, "
           "Query: %d + Target: %d) is smaller than the limit (%d).\n",
           combined_size, query_size, target_size,
           ini->getCUDAMinCombinedSize());
    printf(
        "cuda     : -- GPU support can be forced using -fgpu 1 or by changing "
        "the min. combined size via -mc <limit>.\n");
  }
}

void Acc::upload_molecules(Ini *ini) {
  if (!this->is_using_CUDA()) {
    return;
  }

  // Query sizes
  double *h_query_atoms = ini->getMolQuery()->getAtomsXYZ();
  double *h_query_weight = ini->getMolQuery()->getWeightAtoms();
  double *h_query_radius = ini->getMolQuery()->getRadiusAtoms();
  uint32_t query_bytes = ini->getMolQuery()->getNumAtoms() * sizeof(vec3d);
  uint32_t query_bytes_rem = ini->getMolQuery()->getNumAtoms() * sizeof(double);

  // Async stream
  hipStream_t stream;
  CUDA_CHECK(hipStreamCreate(&stream));

  // Query allocations
  CUDA_CHECK(hipMallocAsync(&this->query_atoms, query_bytes, stream));
  CUDA_CHECK(hipMallocAsync(&this->query_weight, query_bytes_rem, stream));
  CUDA_CHECK(hipMallocAsync(&this->query_radius, query_bytes_rem, stream));
  // Query copies
  CUDA_CHECK(hipMemcpyAsync(this->query_atoms, h_query_atoms, query_bytes,
                             hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(this->query_weight, h_query_weight,
                             query_bytes_rem, hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(this->query_radius, query_radius, query_bytes_rem,
                             hipMemcpyHostToDevice, stream));

  // Target sizes
  double *h_target_atoms = ini->getMolVariable()->getAtomsXYZ();
  double *h_target_weight = ini->getMolVariable()->getWeightAtoms();
  double *h_target_radius = ini->getMolVariable()->getRadiusAtoms();
  uint32_t target_bytes = ini->getMolVariable()->getNumAtoms() * sizeof(vec3d);
  uint32_t target_bytes_rem =
      ini->getMolVariable()->getNumAtoms() * sizeof(double);

  // Target allocations
  CUDA_CHECK(hipMallocAsync(&this->target_atoms, target_bytes, stream));
  CUDA_CHECK(hipMallocAsync(&this->target_weight, target_bytes_rem, stream));
  CUDA_CHECK(hipMallocAsync(&this->target_radius, target_bytes_rem, stream));

  // Target upload
  CUDA_CHECK(hipMemcpyAsync(this->target_atoms, h_target_atoms, target_bytes,
                             hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(this->target_weight, h_target_weight,
                             target_bytes_rem, hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(this->target_radius, h_target_radius,
                             target_bytes_rem, hipMemcpyHostToDevice, stream));

  CUDA_CHECK(hipStreamSynchronize(stream));
  CUDA_CHECK(hipStreamDestroy(stream));
}

// TODO: Keep reducing register usage (currently is 40 for sm_70 - CUDA 11.7)
// 32 or less would be ideal
// use set(CMAKE_CUDA_FLAGS ${CMAKE_CUDA_FLAGS} --ptxas-options=-v) with cmake
__global__ void kernelRotateMol(const vec3d *atoms, uint32_t atoms_len,
                                const vec3d *x, vec3d *new_atoms) {
  uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= atoms_len) {
    return;
  }

  vec3d vector = (x[2] - x[1]).normalized() * sin(x[0].x() * 0.5);
  vec3d temp = atoms[idx] - x[1];

  quatd q1(cos(x[0].x() * 0.5), vector.x(), vector.y(), vector.z());
  quatd atom_position(0.0, temp.x(), temp.y(), temp.z());

  quatd part2 = q1 * atom_position;
  quatd part3 = part2 * q1.conjugate();

  new_atoms[idx] = x[1] + part3.vec();
}

__global__ void kernelMolToNewPosition(vec3d *new_atoms, uint32_t atoms_len,
                                       const vec3d *x) {
  uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= atoms_len) {
    return;
  }

  new_atoms[idx] += x[3];
}

__global__ void kernelPreciseOverlapSameVDW(
    const vec3d *atoms_query, const double *weight_query,
    const double *radius_query, uint32_t query_size, const vec3d *new_atoms,
    const double *weight_target, const double *radius_target,
    uint32_t target_size, double *results) {
  uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;

  if (idx >= query_size || idy >= target_size) {
    return;
  }

  vec3d temp = atoms_query[idx] - new_atoms[idy];
  double Rij2 = temp.squaredNorm();

  double Kij = exp(Rij2 * -0.3731438999881213);
  double Vij = 24.428790199 * Kij;

  results[idy * query_size + idx] =
      weight_query[idx] * weight_target[idy] * Vij;
}

__global__ void kernelPreciseOverlapNotSameVDW(
    const vec3d *atoms_query, const double *weight_query,
    const double *radius_query, uint32_t query_size, const vec3d *new_atoms,
    const double *weight_target, const double *radius_target,
    uint32_t target_size, double *results) {
  uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;

  if (idx >= query_size || idy >= target_size) {
    return;
  }

  vec3d temp = atoms_query[idx] - new_atoms[idy];
  double Rij2 = temp.squaredNorm();

  double alphai = 2.417972471923026 / (radius_query[idx] * radius_target[idy]);
  double alphaj = 2.417972471923026 / (pow(radius_target[idy], 2.0));

  double Kij = exp(-(alphai * alphaj * Rij2) / (alphai + alphaj));
  double Vij = 7.99984656 * Kij * pow((M_PI / (alphai + alphaj)), 1.5);

  results[idy * query_size + idx] =
      weight_query[idx] * weight_target[idy] * Vij;
}

double Acc::ndim_value(const double *x) {
  // Set device
  static std::atomic_uint32_t current_dev = 0;
  CUDA_CHECK(hipSetDevice(current_dev));
  current_dev = (current_dev + 1) % this->max_CUDA_devices;

  // Pre-data
  uint32_t query_atoms_len = INI.getMolQuery()->getNumAtoms();
  uint32_t target_atoms_len = INI.getMolVariable()->getNumAtoms();
  bool SVDW = INI.getSameVanDerWaalsRadius();
  double query_tanimoto = INI.getMolQuery()->tanimoto;
  double target_tanimoto = INI.getMolVariable()->tanimoto;

  uint32_t new_x_bytes = 4 * sizeof(vec3d);
  uint32_t new_atoms_bytes = target_atoms_len * sizeof(vec3d);
  uint32_t VABs_bytes = query_atoms_len * target_atoms_len * sizeof(double);

  // Host
  static thread_local hipStream_t stream;
  static thread_local bool initialized = false;

  static thread_local double *h_new_x;
  static thread_local double *h_VABs;
  // Device
  static thread_local vec3d *d_x;
  static thread_local vec3d *d_new_atoms;
  static thread_local double *d_VABs;

  if (!initialized) {
    initialized = true;
    // Host
    CUDA_CHECK(hipStreamCreate(&stream));
    h_new_x = new double[12];
    h_VABs = new double[query_atoms_len * target_atoms_len];

    // Device
    CUDA_CHECK(hipMallocAsync(&d_x, new_x_bytes, stream));
    CUDA_CHECK(hipMallocAsync(&d_new_atoms, new_atoms_bytes, stream));
    CUDA_CHECK(hipMallocAsync(&d_VABs, VABs_bytes, stream));
  }

  // Data copies host
  h_new_x[0] = x[0];
  h_new_x[1] = x[0];
  memcpy(h_new_x + 2, x, 10 * sizeof(double));

  // Data copies device
  CUDA_CHECK(hipMemcpyAsync(d_x, h_new_x, new_x_bytes, hipMemcpyHostToDevice,
                             stream));

  // Kernel launches
  // Rotate mol.
  uint32_t rotate_block_size = 512;
  uint32_t rotate_actual_grid_size =
      (target_atoms_len + rotate_block_size - 1) / rotate_block_size;

  kernelRotateMol<<<rotate_actual_grid_size, rotate_block_size, 0, stream>>>(
      this->target_atoms, target_atoms_len, d_x, d_new_atoms);

  // MolToNewPosition
  uint32_t position_block_size = 1024;
  uint32_t position_actual_grid_size =
      (target_atoms_len + rotate_block_size - 1) / rotate_block_size;

  kernelMolToNewPosition<<<position_actual_grid_size, position_block_size, 0,
                           stream>>>(d_new_atoms, target_atoms_len, d_x);

  // Overlap
  // ! Big molecule must be always be the query one
  // Kernel are splitted because combined they were using 32 registers.
  // A bigger register usage would have had tanked some performance.
  // Splitting kernels, we assure when same Van der Waals radius are used
  // with other CUDA versions / other devices / whatever, that kernel remains
  // well under 32 registers.

  dim3 precise_block_size(32, 16);
  dim3 precise_actual_grid_size(
      (query_atoms_len + precise_block_size.x - 1) / precise_block_size.x,
      (target_atoms_len + precise_block_size.y - 1) / precise_block_size.y);
  if (SVDW) {
    kernelPreciseOverlapSameVDW<<<precise_actual_grid_size, precise_block_size,
                                  0, stream>>>(
        this->query_atoms, this->query_weight, this->query_radius,
        query_atoms_len, d_new_atoms, this->target_weight, this->target_radius,
        target_atoms_len, d_VABs);
  } else {

    kernelPreciseOverlapNotSameVDW<<<precise_actual_grid_size,
                                     precise_block_size, 0, stream>>>(
        this->query_atoms, this->query_weight, this->query_radius,
        query_atoms_len, d_new_atoms, this->target_weight, this->target_radius,
        target_atoms_len, d_VABs);
  }

  CUDA_CHECK(hipMemcpyAsync(h_VABs, d_VABs, VABs_bytes, hipMemcpyDeviceToHost,
                             stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  double VAB = 0.0;
  for (uint32_t i = 0; i < query_atoms_len * target_atoms_len; i++) {
    VAB += h_VABs[i];
  }

  return (VAB / (target_tanimoto + query_tanimoto - VAB));
}